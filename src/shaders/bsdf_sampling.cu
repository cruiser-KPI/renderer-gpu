#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../math/basic.h"
#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/materialdata.h"


////////////////////////////////////////////////////////////
// Diffuse BSDF (Lambertian)
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_diffuse_reflection(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    // Cosine weighted hemisphere sampling for Lambert material.
    unitSquareToCosineHemisphere(rng2(prd.seed), state.normal, prd.wi, prd.pdf);

    if (prd.pdf <= 0.0f || optix::dot(prd.wi, state.geoNormal) <= 0.0f)
    {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    // This would be the universal implementation for an arbitrary sampling of a diffuse surface.
    // prd.f_over_pdf = parameters.albedo * (M_1_PIf * fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);

    // PERF Since the cosine-weighted hemisphere distribution is a perfect importance-sampling of the Lambert material,
    // the whole term ((M_1_PIf * fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf) is always 1.0f here!
    prd.f_over_pdf = parameters.albedo;

    prd.flags |= FLAG_DIFFUSE; // Direct lighting will be done with multiple importance sampling.
}


RT_CALLABLE_PROGRAM float4 eval_bsdf_diffuse_reflection(MaterialParameter const& parameters, State const& state,
    PerRayData const& prd, float3 const& wiL)
{
    const float3 f   = parameters.albedo * M_1_PIf;
    const float  pdf = fmaxf(0.0f, optix::dot(wiL, state.normal) * M_1_PIf);

    return make_float4(f, pdf);
}

////////////////////////////////////////////////////////////
// Specular BSDF
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    prd.wi = optix::reflect(-prd.wo, state.normal);

    if (optix::dot(prd.wi, state.geoNormal) <= 0.0f) // Do not sample opaque materials below the geometric surface.
    {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    prd.f_over_pdf = parameters.albedo;
    prd.pdf        = 1.0f;
}

// This is actually never reached, because the FLAG_DIFFUSE flag is not set when a specular BSDF is has been sampled.
RT_CALLABLE_PROGRAM float4 eval_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
return make_float4(0.0f);
}

////////////////////////////////////////////////////////////
// Specular reflection and transmission BSDF (with Fresnel)
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_specular_reflection_transmission(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    // Return the current material's absorption coefficient and ior to the integrator to be able to support nested materials.
    prd.absorption_ior = make_float4(parameters.absorption, parameters.ior);

    const float eta = (prd.flags & FLAG_FRONTFACE)
                      ? prd.absorption_ior.w / prd.ior.x
                      : prd.ior.y / prd.absorption_ior.w;

    const float3 R = optix::reflect(-prd.wo, state.normal);

    float reflective = 1.0f;

    if (optix::refract(prd.wi, -prd.wo, state.normal, eta))
    {
        prd.wi = -prd.wo;
        // Total internal reflection will leave this reflection probability at 1.0f.
        reflective = evaluateFresnelDielectric(eta, optix::dot(prd.wo, state.normal));
    }

    const float pseudo = rng(prd.seed);
    if (pseudo < reflective)
    {
        prd.wi = R; // Fresnel reflection or total internal reflection.
    }

    // No Fresnel factor here. The probability to pick one or the other side took care of that.
    prd.f_over_pdf = parameters.albedo;
    prd.pdf        = 1.0f;
}

RT_CALLABLE_PROGRAM float4 eval_bsdf_specular_reflection_transmission(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
  return make_float4(0.0f);
}