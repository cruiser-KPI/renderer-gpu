#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../math/basic.h"
#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/materialdata.h"


////////////////////////////////////////////////////////////
// Math helpers
////////////////////////////////////////////////////////////


RT_FUNCTION float3 world_to_local(const float3 &w, const State& state)
{
    return make_float3(dot(w, state.tangent), dot(w, state.bitangent), dot(w, state.normal));

}

RT_FUNCTION float3 local_to_world(const float3 &w, const State& state)
{
    return make_float3(state.tangent.x * w.x + state.bitangent.x * w.y + state.normal.x * w.z,
                       state.tangent.y * w.x + state.bitangent.y * w.y + state.normal.y * w.z,
                       state.tangent.z * w.x + state.bitangent.z * w.y + state.normal.z * w.z);
}

RT_FUNCTION float cos_theta(const float3 &w) { return w.z;}
RT_FUNCTION float cos2_theta(const float3 &w) { return w.z * w.z; }
RT_FUNCTION float abs_cos_theta(const float3 &w) { return fabsf(w.z); }
RT_FUNCTION float sin2_theta(const float3 &w) { return fmaxf(0.f, 1.f - cos2_theta(w)); }
RT_FUNCTION float sin_theta(const float3 &w) {return sqrtf(sin2_theta(w)); }
RT_FUNCTION float tan_theta(const float3 &w) {return sin_theta(w) / cos_theta(w); }
RT_FUNCTION float tan2_theta(const float3 &w) {return sin2_theta(w) / cos2_theta(w); }
RT_FUNCTION float cos_phi(const float3 &w) {
    float sinTheta = sin_theta(w);
    return (sinTheta == 0) ? 1 : clamp(w.x / sinTheta, -1.f, 1.f);
}
RT_FUNCTION float sin_phi(const float3 &w) {
    float sinTheta = sin_theta(w);
    return (sinTheta == 0) ? 0 : clamp(w.y / sinTheta, -1.f, 1.f);
}
RT_FUNCTION float cos2_phi(const float3 &w) { return cos_phi(w) * cos_phi(w); }
RT_FUNCTION float sin2_phi(const float3 &w) { return sin_phi(w) * sin_phi(w); }

RT_FUNCTION bool same_hemisphere(const float3 &w, const float3 &wp) {
    return w.z * wp.z > 0;
}

RT_FUNCTION float3 spherical_to_cartesian(float sinTheta, float cosTheta, float phi) {
    return make_float3(sinTheta * cosf(phi), sinTheta * sinf(phi), cosTheta);
}

RT_FUNCTION float3 Reflect(const float3 &wo, const float3 &n) {
    return -wo + 2 * dot(wo, n) * n;
}

RT_FUNCTION bool Refract(const float3 &wi, const float3 &n, float eta,
                    float3 *wt) {
    // Compute $\cos \theta_\roman{t}$ using Snell's law
    float cosThetaI = dot(n, wi);
    float sin2ThetaI = fmaxf(0.f, 1 - cosThetaI * cosThetaI);
    float sin2ThetaT = eta * eta * sin2ThetaI;

    // Handle total internal reflection for transmission
    if (sin2ThetaT >= 1) return false;
    float cosThetaT = std::sqrt(1 - sin2ThetaT);
    *wt = eta * -wi + (eta * cosThetaI - cosThetaT) * n;
    return true;
}


RT_FUNCTION float fresnel_dielectric(float cosThetaI, float etaI, float etaT) {
    cosThetaI = clamp(cosThetaI, -1.f, 1.f);
    // Potentially swap indices of refraction
    bool entering = cosThetaI > 0.f;
    if (!entering) {
        float tmp = etaI;
        etaI = etaT;
        etaT = tmp;
        cosThetaI = std::abs(cosThetaI);
    }

    // Compute _cosThetaT_ using Snell's law
    float sinThetaI = sqrtf(fmaxf(0.f, 1.f - cosThetaI * cosThetaI));
    float sinThetaT = etaI / etaT * sinThetaI;

    // Handle total internal reflection
    if (sinThetaT >= 1) return 1;
    float cosThetaT = sqrtf(fmaxf(0.f, 1.f - sinThetaT * sinThetaT));
    float Rparl = ((etaT * cosThetaI) - (etaI * cosThetaT)) /
        ((etaT * cosThetaI) + (etaI * cosThetaT));
    float Rperp = ((etaI * cosThetaI) - (etaT * cosThetaT)) /
        ((etaI * cosThetaI) + (etaT * cosThetaT));
    return (Rparl * Rparl + Rperp * Rperp) / 2;
}

// https://seblagarde.wordpress.com/2013/04/29/memo-on-fresnel-equations/
RT_FUNCTION float3 fresnel_conductor(float cosThetaI, const float3 &etai,
                                     const float3 &etat, const float3 &k) {
    cosThetaI = clamp(cosThetaI, -1.f, 1.f);
    float3 eta = etat / etai;
    float3 etak = k / etai;

    float cosThetaI2 = cosThetaI * cosThetaI;
    float sinThetaI2 = 1.f - cosThetaI2;
    float3 eta2 = eta * eta;
    float3 etak2 = etak * etak;

    float3 t0 = eta2 - etak2 - sinThetaI2;
    float3 a2plusb2 = sqrt(t0 * t0 + 4 * eta2 * etak2);
    float3 t1 = a2plusb2 + cosThetaI2;
    float3 a = sqrt(0.5f * (a2plusb2 + t0));
    float3 t2 = 2.f * cosThetaI * a;
    float3 Rs = (t1 - t2) / (t1 + t2);

    float3 t3 = cosThetaI2 * a2plusb2 + sinThetaI2 * sinThetaI2;
    float3 t4 = t2 * sinThetaI2;
    float3 Rp = Rs * (t3 - t4) / (t3 + t4);

    return 0.5 * (Rp + Rs);
}


RT_FUNCTION void roughness_to_alpha(float r, float aniso, float *alphax, float *alphay)
{
    r = fmaxf(r, 1e-3f);
    aniso = clamp(aniso, -0.99f, 0.99f);

    if (aniso < 0.0) {
        *alphax = r / (1.f + aniso);
        *alphay = r * (1.f + aniso);
    }
    else {
        *alphax = r * (1.f - aniso);
        *alphay = r / (1.f - aniso);
    }

    // TODO find out why we don't need to remap roughness
//    float x = log(roughness);
//    return 1.62142f + 0.819955f * x + 0.1734f * x * x + 0.0171201f * x * x * x +
//        0.000640711f * x * x * x * x;
}

RT_FUNCTION float ggx_aniso_d(const float3 &wh, float alphax, float alphay)
{
    float tan2Theta = tan2_theta(wh);
    if (isinf(tan2Theta)) return 0;
    const float cos4Theta = cos2_theta(wh) * cos2_theta(wh);
    float e = (cos2_phi(wh) / (alphax * alphax) + sin2_phi(wh) / (alphay * alphay)) * tan2Theta;
    return 1 / (M_PIf * alphax * alphay * cos4Theta * (1 + e) * (1 + e));
}

RT_FUNCTION float ggx_aniso_lambda(const float3 &w, float alphax, float alphay)
{
    float absTanTheta = fabsf(tan_theta(w));
    if (isinf(absTanTheta)) return 0;
    // Compute _alpha_ for direction _w_
    float alpha_hat = sqrtf(cos2_phi(w) * alphax * alphax + sin2_phi(w) * alphay * alphay);
    float alpha2Tan2Theta = (alpha_hat * absTanTheta) * (alpha_hat * absTanTheta);
    return (-1 + sqrtf(1.f + alpha2Tan2Theta)) / 2;
}

RT_FUNCTION float ggx_aniso_g(const float3 &wo, const float3 &wi, float alphax, float alphay)
{
    return 1.f / (1.f + ggx_aniso_lambda(wo, alphax, alphay) + ggx_aniso_lambda(wi, alphax, alphay));
}

RT_FUNCTION float ggx_aniso_pdf(const float3 &wo, const float3 &wh, float alphax, float alphay)
{
    float G1 = 1.f / (1.f + ggx_aniso_lambda(wo, alphax, alphay));
    return ggx_aniso_d(wh, alphax, alphay) * G1 * abs(dot(wo, wh)) / abs_cos_theta(wo);
}

RT_FUNCTION void ggx_aniso_sample11(float cosTheta, float U1, float U2,
                                    float *slope_x, float *slope_y) {
    // special case (normal incidence)
    if (cosTheta > .9999) {
        float r = sqrt(U1 / (1 - U1));
        float phi = 6.28318530718f * U2;
        *slope_x = r * cos(phi);
        *slope_y = r * sin(phi);
        return;
    }

    float sinTheta = sqrtf(fmaxf(0.f, 1.f - cosTheta * cosTheta));
    float tanTheta = sinTheta / cosTheta;
    float a = 1 / tanTheta;
    float G1 = 2 / (1 + sqrtf(1.f + 1.f / (a * a)));

    // sample slope_x
    float A = 2 * U1 / G1 - 1;
    float tmp = 1.f / (A * A - 1.f);
    if (tmp > 1e10) tmp = 1e10;
    float B = tanTheta;
    float D = sqrtf(fmaxf(B * B * tmp * tmp - (A * A - B * B) * tmp, .0f));
    float slope_x_1 = B * tmp - D;
    float slope_x_2 = B * tmp + D;
    *slope_x = (A < 0 || slope_x_2 > 1.f / tanTheta) ? slope_x_1 : slope_x_2;

    // sample slope_y
    float S;
    if (U2 > 0.5f) {
        S = 1.f;
        U2 = 2.f * (U2 - .5f);
    } else {
        S = -1.f;
        U2 = 2.f * (.5f - U2);
    }
    float z = (U2 * (U2 * (U2 * 0.27385f - 0.73369f) + 0.46341f)) /
            (U2 * (U2 * (U2 * 0.093073f + 0.309420f) - 1.000000f) + 0.597999f);
    *slope_y = S * z * std::sqrt(1.f + *slope_x * *slope_x);
}


RT_FUNCTION float3 ggx_aniso_sample_wh(const float3 &wo, const float2 &u, float alphax, float alphay)
{
    bool flip = wo.z < 0;
    float3 wi = (flip) ? -wo : wo;

    // 1. stretch wi
    float3 wiStretched = normalize(make_float3(alphax * wi.x, alphay * wi.y, wi.z));

    // 2. simulate P22_{wi}(x_slope, y_slope, 1, 1)
    float slope_x, slope_y;
    ggx_aniso_sample11(cos_theta(wiStretched), u.x, u.y, &slope_x, &slope_y);

    // 3. rotate
    float tmp = cos_phi(wiStretched) * slope_x - sin_phi(wiStretched) * slope_y;
    slope_y = sin_phi(wiStretched) * slope_x + cos_phi(wiStretched) * slope_y;
    slope_x = tmp;

    // 4. unstretch
    slope_x = alphax * slope_x;
    slope_y = alphay * slope_y;

    // 5. compute normal
    float3 wh = normalize(make_float3(-slope_x, -slope_y, 1.f));
    return (flip) ? -wh : wh;
}




////////////////////////////////////////////////////////////
// Diffuse BSDF (Lambertian)
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_diffuse_reflection(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    // Cosine weighted hemisphere sampling for Lambert material.
    unitSquareToCosineHemisphere(rng2(prd.seed), state.normal, prd.wi, prd.pdf);

    if (prd.pdf <= 0.0f || optix::dot(prd.wi, state.geoNormal) <= 0.0f)
    {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    prd.f_over_pdf = parameters.albedo;
}


RT_CALLABLE_PROGRAM float4 eval_bsdf_diffuse_reflection(MaterialParameter const& parameters, State const& state,
    PerRayData const& prd, float3 const& wiL)
{
    const float3 f   = parameters.albedo * M_1_PIf;
    const float  pdf = fmaxf(0.0f, optix::dot(wiL, state.normal) * M_1_PIf);

    return make_float4(f, pdf);
}


////////////////////////////////////////////////////////////
// Glossy bsdf (with Fresnel)
////////////////////////////////////////////////////////////


RT_CALLABLE_PROGRAM float4 eval_bsdf_glossy(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
    float3 wo = world_to_local(prd.wo, state), wi = world_to_local(wiL, state);
    float cosThetaO = abs_cos_theta(wo), cosThetaI = abs_cos_theta(wi);
    float3 wh = wi + wo;
    // Handle degenerate cases for microfacet reflection
    if ((cosThetaI == 0 || cosThetaO == 0) || (wh.x == 0 && wh.y == 0 && wh.z == 0) ||
        !same_hemisphere(wo, wi))
        return make_float4(0.f);
    wh = normalize(wh);
//    float F = fresnel_dielectric(dot(wi, wh), 1, parameters.ior);

    float alphax, alphay;
    roughness_to_alpha(parameters.roughness, parameters.anisotropy, &alphax, &alphay);
    float3 f = parameters.albedo * ggx_aniso_d(wh, alphax, alphay) *
        ggx_aniso_g(wo, wi, alphax, alphay) / (4 * cosThetaI * cosThetaO);
    float pdf = ggx_aniso_pdf(wo, wh, alphax, alphay) / (4 * dot(wo, wh));

    return make_float4(f, pdf);
}

RT_CALLABLE_PROGRAM void sample_bsdf_glossy(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    float3 wo = world_to_local(prd.wo, state);
    if (wo.z == 0) {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    float alphax, alphay;
    roughness_to_alpha(parameters.roughness, parameters.anisotropy, &alphax, &alphay);
    float3 wh = ggx_aniso_sample_wh(wo, rng2(prd.seed), alphax, alphay);
    float3 wi = normalize(Reflect(wo, wh));

    if (!same_hemisphere(wo, wi)) {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    prd.wi = local_to_world(wi, state);
    float4 bsdf_val = eval_bsdf_glossy(parameters, state, prd, prd.wi);
    prd.pdf = bsdf_val.w;
    prd.f_over_pdf = make_float3(bsdf_val) * fabsf(dot(prd.wi, state.normal)) / prd.pdf;

    if (prd.pdf <= 0.0f || dot(prd.wi, state.geoNormal) <= 0.0f) {
        prd.flags |= FLAG_TERMINATE;
        return;
    }


}

////////////////////////////////////////////////////////////
// Refraction bsdf (with Fresnel)
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM float4 eval_bsdf_refraction(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
    float3 wo = world_to_local(prd.wo, state), wi = world_to_local(wiL, state);
    float cosThetaO = abs_cos_theta(wo), cosThetaI = abs_cos_theta(wi);

    if ((cosThetaI == 0 || cosThetaO == 0) || same_hemisphere(wo, wi))
        return make_float4(0.f);

    float eta = cos_theta(wo) > 0 ? (parameters.ior / 1.f) : (1.f / parameters.ior);
    float3 wh = normalize(wo + wi * eta);
    if (wh.z < 0) wh = -wh;

    // TODO think what to do with fresnel in microfacet models
//    float F = fresnel_dielectric(dot(wo, wh), 1, parameters.ior);
    float sqrtDenom = dot(wo, wh) + eta * dot(wi, wh);

    float alphax, alphay;
    roughness_to_alpha(parameters.roughness, parameters.anisotropy, &alphax, &alphay);
    float3 f =  parameters.albedo * fabsf(ggx_aniso_d(wh, alphax, alphay) *
        ggx_aniso_g(wo, wi, alphax, alphay) * eta * eta * fabsf(dot(wi, wh)) * fabsf(dot(wo, wh))
        / (cosThetaI * cosThetaO * sqrtDenom * sqrtDenom));

    float dwh_dwi = fabsf((eta * eta * dot(wi, wh)) / (sqrtDenom * sqrtDenom));
    float pdf = ggx_aniso_pdf(wo, wh, alphax, alphay) * dwh_dwi;

    return make_float4(f, pdf);

}

RT_CALLABLE_PROGRAM void sample_bsdf_refraction(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    float3 wo = world_to_local(prd.wo, state);
    if (wo.z == 0) {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    float alphax, alphay;
    roughness_to_alpha(parameters.roughness, parameters.anisotropy, &alphax, &alphay);
    float3 wh = ggx_aniso_sample_wh(wo, rng2(prd.seed), alphax, alphay);
    float eta = cos_theta(wo) > 0 ? (1.f / parameters.ior) : (parameters.ior / 1.f);
    float3 wi;
    if (!Refract(wo, wh, eta, &wi)) {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    prd.wi = local_to_world(wi, state);
    float4 bsdf_val = eval_bsdf_refraction(parameters, state, prd, prd.wi);
    prd.pdf = bsdf_val.w;
    prd.f_over_pdf = make_float3(bsdf_val) * fabsf(dot(prd.wi, state.normal)) / prd.pdf;
}


RT_CALLABLE_PROGRAM float4 eval_bsdf_glass(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
    float F = fresnel_dielectric(dot(wiL, state.normal), 1, parameters.ior);
    float4 res = make_float4(0.f);
    unsigned int seed = prd.seed;
    if (rng(seed) < F){
        res = eval_bsdf_glossy(parameters, state, prd, wiL);
        res = make_float4(make_float3(res) * F, res.w / F);
    }
    else {
        res = eval_bsdf_refraction(parameters, state, prd, wiL);
        F = 1 - F;
        res = make_float4(make_float3(res) * F, res.w / F);
    }
    return res;
}


RT_CALLABLE_PROGRAM void sample_bsdf_glass(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    float F = fresnel_dielectric(dot(prd.wo, state.normal), 1, parameters.ior);
    if (rng(prd.seed) < F){
        sample_bsdf_glossy(parameters, state, prd);
        prd.pdf /= F;
        prd.f_over_pdf *= F;
    }
    else {
        sample_bsdf_refraction(parameters, state, prd);
        prd.pdf /= (1 - F);
        prd.f_over_pdf *= (1 - F);
    }
}