#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/materialdata.h"

////////////////////////////////////////////////////////////
// Helper functions
////////////////////////////////////////////////////////////

// This function evaluates a Fresnel dielectric function when the transmitting cosine ("cost")
// is unknown and the incident index of refraction is assumed to be 1.0f.
// \param et     The transmitted index of refraction.
// \param costIn The cosine of the angle between the incident direction and normal direction.
RT_FUNCTION float evaluateFresnelDielectric(const float et, const float cosIn)
{
    const float cosi = fabsf(cosIn);

    float sint = 1.0f - cosi * cosi;
    sint = (0.0f < sint) ? sqrtf(sint) / et : 0.0f;

    // Handle total internal reflection.
    if (1.0f < sint)
    {
        return 1.0f;
    }

    float cost = 1.0f - sint * sint;
    cost = (0.0f < cost) ? sqrtf(cost) : 0.0f;

    const float et_cosi = et * cosi;
    const float et_cost = et * cost;

    const float rPerpendicular = (cosi - et_cost) / (cosi + et_cost);
    const float rParallel      = (et_cosi - cost) / (et_cosi + cost);

    const float result = (rParallel * rParallel + rPerpendicular * rPerpendicular) * 0.5f;

    return (result <= 1.0f) ? result : 1.0f;
}

RT_FUNCTION void alignVector(float3 const& axis, float3& w)
{
    // Align w with axis.
    const float s = copysign(1.0f, axis.z);
    w.z *= s;
    const float3 h = make_float3(axis.x, axis.y, axis.z + s);
    const float  k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
    w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const& axis, float3& w, float& pdf)
{
    // Choose a point on the hemisphere about +z
    const float theta = 2.0f * M_PIf * sample.x;
    const float r = sqrtf(sample.y);
    w.x = r * cosf(theta);
    w.y = r * sinf(theta);
    w.z = 1.0f - w.x * w.x - w.y * w.y;
    w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

    pdf = w.z * M_1_PIf;

    // Align with axis.
    alignVector(axis, w);
}

////////////////////////////////////////////////////////////
// Diffuse BSDF (Lambertian)
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_diffuse_reflection(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    // Cosine weighted hemisphere sampling for Lambert material.
    unitSquareToCosineHemisphere(rng2(prd.seed), state.normal, prd.wi, prd.pdf);

    if (prd.pdf <= 0.0f || optix::dot(prd.wi, state.geoNormal) <= 0.0f)
    {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    // This would be the universal implementation for an arbitrary sampling of a diffuse surface.
    // prd.f_over_pdf = parameters.albedo * (M_1_PIf * fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf);

    // PERF Since the cosine-weighted hemisphere distribution is a perfect importance-sampling of the Lambert material,
    // the whole term ((M_1_PIf * fabsf(optix::dot(prd.wi, state.normal)) / prd.pdf) is always 1.0f here!
    prd.f_over_pdf = parameters.albedo;

    prd.flags |= FLAG_DIFFUSE; // Direct lighting will be done with multiple importance sampling.
}

// The parameter wiL is the lightSample.direction (direct lighting), not the next ray segment's direction prd.wi (indirect lighting).
RT_CALLABLE_PROGRAM float4 eval_bsdf_diffuse_reflection(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
const float3 f   = parameters.albedo * M_1_PIf;
const float  pdf = fmaxf(0.0f, optix::dot(wiL, state.normal) * M_1_PIf);

return make_float4(f, pdf);
}

////////////////////////////////////////////////////////////
// Specular BSDF
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    prd.wi = optix::reflect(-prd.wo, state.normal);

    if (optix::dot(prd.wi, state.geoNormal) <= 0.0f) // Do not sample opaque materials below the geometric surface.
    {
        prd.flags |= FLAG_TERMINATE;
        return;
    }

    prd.f_over_pdf = parameters.albedo;
    prd.pdf        = 1.0f;
}

// This is actually never reached, because the FLAG_DIFFUSE flag is not set when a specular BSDF is has been sampled.
RT_CALLABLE_PROGRAM float4 eval_bsdf_specular_reflection(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
return make_float4(0.0f);
}

////////////////////////////////////////////////////////////
// Specular reflection and transmission BSDF (with Fresnel)
////////////////////////////////////////////////////////////

RT_CALLABLE_PROGRAM void sample_bsdf_specular_reflection_transmission(MaterialParameter const& parameters, State const& state, PerRayData& prd)
{
    // Return the current material's absorption coefficient and ior to the integrator to be able to support nested materials.
    prd.absorption_ior = make_float4(parameters.absorption, parameters.ior);

    // Need to figure out here which index of refraction to use if the ray is already inside some refractive medium.
    // This needs to happen with the original FLAG_FRONTFACE condition to find out from which side of the geometry we're looking!
    // ior.xy are the current volume's IOR and the surrounding volume's IOR.
    // Thin-walled materials have no volume, always use the frontface eta for them!
    const float eta = (prd.flags & (FLAG_FRONTFACE | FLAG_THINWALLED))
                      ? prd.absorption_ior.w / prd.ior.x
                      : prd.ior.y / prd.absorption_ior.w;

    const float3 R = optix::reflect(-prd.wo, state.normal);

    float reflective = 1.0f;

    if (optix::refract(prd.wi, -prd.wo, state.normal, eta))
    {
        if (prd.flags & FLAG_THINWALLED)
        {
            prd.wi = -prd.wo; // Straight through, no volume.
        }
        // Total internal reflection will leave this reflection probability at 1.0f.
        reflective = evaluateFresnelDielectric(eta, optix::dot(prd.wo, state.normal));
    }

    const float pseudo = rng(prd.seed);
    if (pseudo < reflective)
    {
        prd.wi = R; // Fresnel reflection or total internal reflection.
    }
    else if (!(prd.flags & FLAG_THINWALLED)) // Only non-thinwalled materials have a volume and transmission events.
    {
        prd.flags |= FLAG_TRANSMISSION;
    }

    // No Fresnel factor here. The probability to pick one or the other side took care of that.
    prd.f_over_pdf = parameters.albedo;
    prd.pdf        = 1.0f;
}

RT_CALLABLE_PROGRAM float4 eval_bsdf_specular_reflection_transmission(MaterialParameter const& parameters, State const& state, PerRayData const& prd, float3 const& wiL)
{
  return make_float4(0.0f);
}