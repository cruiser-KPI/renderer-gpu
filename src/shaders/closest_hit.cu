#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/materialdata.h"
#include "../core/lightdata.h"
#include "../math/basic.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance,);
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, materialIndex, , ); // Per Material index into the sysMaterialParameters array.

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int, sysNumLights, , );     // PERF Used many times and faster to read than sysLightDefinitions.size().

rtBuffer< rtCallableProgramId<void(float3 const& point, const float2 sample, LightSample& lightSample)> >
    sysSampleLight;

rtBuffer<rtCallableProgramId<void(MaterialParameter const &parameters, State const &state, PerRayData &prd)>>
    sysSampleBSDF;

rtBuffer<rtCallableProgramId<float4(MaterialParameter const& parameters, State const &state, PerRayData const& prd,
    float3 const &wiL)> > sysEvalBSDF;

RT_PROGRAM void closest_hit()
{
    State state; // All in world space coordinates!
    state.geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
    state.normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
    state.texcoord  = varTexCoord;

    thePrd.pos = theRay.origin + theRay.direction * theIntersectionDistance;
    thePrd.distance = theIntersectionDistance;

    thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, state.geoNormal)) ? FLAG_FRONTFACE : 0;

    if ((thePrd.flags & FLAG_FRONTFACE) == 0)
    {
        state.geoNormal = -state.geoNormal;
        state.normal = -state.normal;
    }

    thePrd.radiance = make_float3(0.0f);
    MaterialParameter parameters = sysMaterialParameters[materialIndex];
    if (parameters.textureID != RT_TEXTURE_ID_NULL)
    {
        const float3 texColor = make_float3(optix::rtTex2D<float4>(parameters.textureID,
            parameters.textureScale * state.texcoord.x, parameters.textureScale * state.texcoord.y));
        parameters.albedo *= texColor;
    }

    thePrd.f_over_pdf = make_float3(0.0f);
    thePrd.pdf = 0.0f;
    thePrd.flags = (thePrd.flags & ~FLAG_DIFFUSE) | parameters.flags;

    sysSampleBSDF[parameters.indexBSDF](parameters, state, thePrd);

    if ((thePrd.flags & FLAG_DIFFUSE) && 0 < sysNumLights) {
        const float2 sample = rng2(thePrd.seed);

        LightSample lightSample;
        lightSample.index = optix::clamp(static_cast<int>(
            floorf(rng(thePrd.seed) * sysNumLights)), 0, sysNumLights - 1);

        const LightType lightType = sysLightDefinitions[lightSample.index].type;
        sysSampleLight[lightType](thePrd.pos, sample, lightSample);
        if (0.0f < lightSample.pdf)
        {
            const float4 bsdf_pdf = sysEvalBSDF[parameters.indexBSDF](parameters, state, thePrd, lightSample.direction);

            if (0.0f < bsdf_pdf.w && isNotNull(make_float3(bsdf_pdf))) {

                PerRayData_shadow prdShadow;
                prdShadow.visible = true;

                optix::Ray ray = optix::make_Ray(thePrd.pos,
                                                 lightSample.direction,
                                                 1,
                                                 sysSceneEpsilon,
                                                 lightSample.distance - sysSceneEpsilon); // Shadow ray.
                rtTrace(sysTopObject, ray, prdShadow);

                if (prdShadow.visible) {
                    const float misWeight = powerHeuristic(lightSample.pdf, bsdf_pdf.w);

                    thePrd.radiance += make_float3(bsdf_pdf) * lightSample.emission * float(sysNumLights) *
                        (misWeight * optix::dot(lightSample.direction, state.normal) / lightSample.pdf);
                }
            }
        }
    }
}