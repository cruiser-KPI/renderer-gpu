#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/materialdata.h"
#include "../core/lightdata.h"
#include "../math/basic.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );

rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance,);
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
//rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, materialIndex, , ); // Per Material index into the sysMaterialParameters array.

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int, sysNumLights, , );     // PERF Used many times and faster to read than sysLightDefinitions.size().

rtBuffer< rtCallableProgramId<void(float3 const& point, const float2 sample, LightSample& lightSample)> >
    sysSampleLight;

rtBuffer<rtCallableProgramId<void(MaterialParameter const &parameters, State const &state, PerRayData &prd)>>
    sysSampleBSDF;

rtBuffer<rtCallableProgramId<float4(MaterialParameter const& parameters, State const &state, PerRayData const& prd,
    float3 const &wiL)> > sysEvalBSDF;

// Helper functions for sampling a cosine weighted hemisphere distribution as needed for the Lambert shading model.

RT_FUNCTION void alignVector(float3 const &axis, float3 &w)
{
    // Align w with axis.
    const float s = copysign(1.0f, axis.z);
    w.z *= s;
    const float3 h = make_float3(axis.x, axis.y, axis.z + s);
    const float k = optix::dot(w, h) / (1.0f + fabsf(axis.z));
    w = k * h - w;
}

RT_FUNCTION void unitSquareToCosineHemisphere(const float2 sample, float3 const &axis, float3 &w, float &pdf)
{
    // Choose a point on the local hemisphere coordinates about +z.
    const float theta = 2.0f * M_PIf * sample.x;
    const float r = sqrtf(sample.y);
    w.x = r * cosf(theta);
    w.y = r * sinf(theta);
    w.z = 1.0f - w.x * w.x - w.y * w.y;
    w.z = (0.0f < w.z) ? sqrtf(w.z) : 0.0f;

    pdf = w.z * M_1_PIf;

    // Align with axis.
    alignVector(axis, w);
}

RT_PROGRAM void closest_hit()
{
    State state; // All in world space coordinates!
    state.geoNormal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
    state.normal = optix::normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
    state.texcoord  = varTexCoord;

    thePrd.pos = theRay.origin
        + theRay.direction * theIntersectionDistance; // Advance the path to the hit position in world coordinates.
    thePrd.distance =
        theIntersectionDistance; // Return the current path segment distance, needed for absorption calculations in the integrator.

    // Explicitly include edge-on cases as frontface condition!
    // Keeps the material stack from overflowing at silhouttes.
    // Prevents that silhouettes of thin-walled materials use the backface material.
    // Using the true geometry normal attribute as originally defined on the frontface!
    thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, state.geoNormal)) ? FLAG_FRONTFACE : 0;

    if ((thePrd.flags & FLAG_FRONTFACE) == 0) // Looking at the backface?
    {
        // Means geometric normal and shading normal are always defined on the side currently looked at.
        // This gives the backfaces of opaque BSDFs a defined result.
        state.geoNormal = -state.geoNormal;
        state.normal = -state.normal;
        // Do not recalculate the frontface condition!
    }

    // A material system with support for arbitrary mesh lights would evaluate its emission here.
    // But since only parallelogram area lights are supported, those get a dedicated closest hit program to simplify this demo.
    thePrd.radiance = make_float3(0.0f);

    MaterialParameter parameters = sysMaterialParameters[materialIndex];

    if (parameters.textureID != RT_TEXTURE_ID_NULL)
    {
        const float3 texColor = make_float3(optix::rtTex2D<float4>(parameters.textureID,
            parameters.textureScale * state.texcoord.x, parameters.textureScale * state.texcoord.y));

        // Modulate the incoming color with the texture.
        parameters.albedo *= texColor;               // linear color, resp. if the texture has been uint8 and readmode set to use sRGB, then sRGB.
        //parameters.albedo *= powf(texColor, 2.2f); // sRGB gamma correction done manually.
    }

    // Start fresh with the next BSDF sample.  (Either of these values remaining zero is an end-of-path condition.)
    thePrd.f_over_pdf = make_float3(0.0f);
    thePrd.pdf = 0.0f;

    // Only the last diffuse hit is tracked for multiple importance sampling of implicit light hits.
    thePrd.flags = (thePrd.flags & ~FLAG_DIFFUSE)
        | parameters.flags; // FLAG_THINWALLED can be set directly from the material parameters.

    sysSampleBSDF[parameters.indexBSDF](parameters, state, thePrd);

    // Direct lighting if the sampled BSDF was diffuse and any light is in the scene.
    if ((thePrd.flags & FLAG_DIFFUSE) && 0 < sysNumLights) {
        const float2
            sample = rng2(thePrd.seed); // Use higher dimension samples for the position. (Irrelevant for the LCG).

        LightSample
        lightSample; // Sample one of many lights.

        // The caller picks the light to sample. Make sure the index stays in the bounds of the sysLightDefinitions array.
        lightSample.index =
            optix::clamp(static_cast<int>(floorf(rng(thePrd.seed) * sysNumLights)), 0, sysNumLights - 1);

        const LightType lightType = sysLightDefinitions[lightSample.index].type;

        sysSampleLight[lightType](thePrd.pos, sample, lightSample);

        if (0.0f < lightSample.pdf) // Useful light sample?
        {
            // Evaluate the BSDF in the light sample direction. Normally cheaper than shooting rays.
            // Returns BSDF f in .xyz and the BSDF pdf in .w
            const float4 bsdf_pdf = sysEvalBSDF[parameters.indexBSDF](parameters, state, thePrd, lightSample.direction);

            if (0.0f < bsdf_pdf.w && isNotNull(make_float3(bsdf_pdf))) {
                // Do the visibility check of the light sample.
                PerRayData_shadow prdShadow;

                prdShadow.visible = true; // Initialize for miss.

                // Note that the sysSceneEpsilon is applied on both sides of the shadow ray [t_min, t_max] interval
                // to prevent self intersections with the actual light geometry in the scene!
                optix::Ray ray = optix::make_Ray(thePrd.pos,
                                                 lightSample.direction,
                                                 1,
                                                 sysSceneEpsilon,
                                                 lightSample.distance - sysSceneEpsilon); // Shadow ray.
                rtTrace(sysTopObject, ray, prdShadow);

                if (prdShadow.visible) {
                    if (thePrd.flags
                        & FLAG_VOLUME) // Supporting nested materials includes having lights inside a volume.
                    {
                        // Calculate the transmittance along the light sample's distance in case it's inside a volume.
                        // The light must be in the same volume or it would have been shadowed!
                        lightSample.emission *= expf(-lightSample.distance * thePrd.extinction);
                    }

                    const float misWeight = powerHeuristic(lightSample.pdf, bsdf_pdf.w);

                    thePrd.radiance += make_float3(bsdf_pdf) * lightSample.emission * float(sysNumLights) *
                        (misWeight * optix::dot(lightSample.direction, state.normal) / lightSample.pdf);
                }
            }
        }
    }
}