#include "hip/hip_runtime.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
#include <optix_world.h>
using namespace optix;

#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/materialdata.h"
#include "../core/lightdata.h"
#include "../math/basic.h"

// Context global variables provided by the renderer system.
rtDeclareVariable(rtObject, sysTopObject, , );
rtDeclareVariable(float, sysSceneEpsilon, , );

// Semantic variables.
rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(float, theIntersectionDistance, rtIntersectionDistance,);
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal, attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

// Material parameter definition.
rtBuffer<MaterialParameter> sysMaterialParameters; // Context global buffer with an array of structures of MaterialParameter.
rtDeclareVariable(int, materialIndex, , ); // Per Material index into the sysMaterialParameters array.

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int, sysNumLights, , );     // PERF Used many times and faster to read than sysLightDefinitions.size().

rtBuffer< rtCallableProgramId<void(float3 const& point, const float2 sample, LightSample& lightSample)> >
    sysSampleLight;

rtBuffer<rtCallableProgramId<void(MaterialParameter const &parameters, State const &state, PerRayData &prd)>>
    sysSampleBSDF;

rtBuffer<rtCallableProgramId<float4(MaterialParameter const& parameters, State const &state, PerRayData const& prd,
    float3 const &wiL)> > sysEvalBSDF;

RT_PROGRAM void closest_hit()
{
    if (sysNumLights == 0)
        return;

    MaterialParameter parameters = sysMaterialParameters[materialIndex];
    float mixFactor = 1.f;
    while (parameters.indexBSDF == MaterialType::MIX) // handle mix materials
    {
        // TODO fix mix factor (using ior for mix factor)
        // ior is used here for mix factor

        parameters.ior = clamp(parameters.ior, 0.01f, 0.99f);

        if (rng(thePrd.seed) < parameters.ior) {
            mixFactor = parameters.ior;
            parameters = sysMaterialParameters[materialIndex + 1];
        }
        else {
            mixFactor = (1 - parameters.ior);
            parameters = sysMaterialParameters[materialIndex + 2];
        }
    }

    State state; // All in world space coordinates!
    state.geoNormal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varGeoNormal));
    state.normal = normalize(rtTransformNormal(RT_OBJECT_TO_WORLD, varNormal));
    if (parameters.rotation != 0)
        // rotate tangent for anisotropic materials
        state.tangent = make_float3(normalize(
            Matrix4x4::rotate(parameters.rotation * 2 * M_PIf, state.normal) * make_float4(RT_OBJECT_TO_WORLD * varTangent)));
    else
        state.tangent = normalize(RT_OBJECT_TO_WORLD * varTangent);
    state.texcoord  = varTexCoord;

    thePrd.flags |= (0.0f <= optix::dot(thePrd.wo, state.geoNormal)) ? FLAG_FRONTFACE : 0;

    if ((thePrd.flags & FLAG_FRONTFACE) == 0)
    {
        state.geoNormal = -state.geoNormal;
        state.normal = -state.normal;
    }
    state.bitangent = cross(state.normal, state.tangent);

    thePrd.pos = theRay.origin + theRay.direction * theIntersectionDistance;
    thePrd.distance = theIntersectionDistance;

    thePrd.radiance = make_float3(0.0f);
    if (parameters.textureID != RT_TEXTURE_ID_NULL)
    {
        const float3 texColor = make_float3(optix::rtTex2D<float4>(parameters.textureID,
                                                                   parameters.textureScale * state.texcoord.x, parameters.textureScale * state.texcoord.y));
        parameters.albedo *= texColor;
    }
    thePrd.f_over_pdf = make_float3(0.0f);
    thePrd.pdf = 0.0f;
    thePrd.flags = parameters.flags | FLAG_PATH;

    // --- importance sample light source
    const float2 sample = rng2(thePrd.seed);

    LightSample lightSample;
    lightSample.index = optix::clamp(static_cast<int>(
                                         floorf(rng(thePrd.seed) * sysNumLights)), 0, sysNumLights - 1);


    const LightType lightType = sysLightDefinitions[lightSample.index].type;
    sysSampleLight[lightType](thePrd.pos, sample, lightSample);
    if (0.0f < lightSample.pdf) {
        // handle delta lights
        float4 bsdf_pdf = sysEvalBSDF[parameters.indexBSDF](parameters, state, thePrd, lightSample.direction);

        if (0.0f < bsdf_pdf.w && isNotNull(make_float3(bsdf_pdf))) {

            PerRayData_shadow prdShadow;
            prdShadow.visible = true;

            optix::Ray ray = optix::make_Ray(thePrd.pos,
                                             lightSample.direction,
                                             1,
                                             sysSceneEpsilon,
                                             lightSample.distance - sysSceneEpsilon); // Shadow ray.
            rtTrace(sysTopObject, ray, prdShadow);

            if (prdShadow.visible) {
                bsdf_pdf = make_float4(make_float3(bsdf_pdf) * mixFactor, bsdf_pdf.w / mixFactor);

                // don't importance sample delta lights
                if (lightType == DIRECTIONAL || lightType == POINT) {
                    // TODO solve dark spot problem in transparent material for directional lights
                    thePrd.radiance += make_float3(bsdf_pdf) * lightSample.emission * float(sysNumLights) *
                        (dot(lightSample.direction, state.normal) / lightSample.pdf);
                }
                else {
                    const float misWeight = powerHeuristic(lightSample.pdf, bsdf_pdf.w);

                    thePrd.radiance += make_float3(bsdf_pdf) * lightSample.emission * float(sysNumLights) *
                        (misWeight * dot(lightSample.direction, state.normal) / lightSample.pdf);
                }
            }
        }

    }

    // --- sample BSDF to find next ray direction
    sysSampleBSDF[parameters.indexBSDF](parameters, state, thePrd);
    thePrd.pdf /= mixFactor;
    thePrd.f_over_pdf *= mixFactor;

}