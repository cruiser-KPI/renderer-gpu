#include "hip/hip_runtime.h"

#include "../utils/config.h"
#include <optix.h>
#include <optixu/optixu_math_namespace.h>
using namespace optix;

rtBuffer<float4, 2> sysOutputBuffer; // RGBA32F

rtDeclareVariable(uint2, theLaunchIndex, rtLaunchIndex, );
rtDeclareVariable(uint2, tileOffset, , );

RT_PROGRAM void exception()
{
#ifdef USE_DEBUG_EXCEPTIONS
  const unsigned int code = rtGetExceptionCode();
  const uint2 tileLaunchIndex = tileOffset + theLaunchIndex;
  if (RT_EXCEPTION_USER <= code)
  {
    rtPrintf("User exception %d at (%d, %d)\n", code - RT_EXCEPTION_USER, tileLaunchIndex.x, tileLaunchIndex.y);
  }
  else
  {
    rtPrintf("Exception code 0x%X at (%d, %d)\n", code, tileLaunchIndex.x, tileLaunchIndex.y);
  }
  rtPrintExceptionDetails();
  // RGBA32F super magenta as error color (makes sure this isn't accumulated away in a progressive renderer).

  sysOutputBuffer[tileLaunchIndex] = make_float4(1000000.0f, 0.0f, 1000000.0f, 1.0f);
#endif
}
