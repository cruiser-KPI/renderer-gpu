#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../utils/config.h"
#include "../core/perraydata.h"
#include "../core/lightdata.h"
#include "../math/basic.h"

rtBuffer<LightDefinition> sysLightDefinitions;
rtDeclareVariable(int,    sysNumLights, , );
rtDeclareVariable(float,  sysEnvironmentRotation, , );

// Note that all light sampling routines return lightSample.direction and lightSample.distance in world space!

RT_CALLABLE_PROGRAM void sample_environment_light(float3 const& point, const float2 sample, LightSample& lightSample)
{
    LightDefinition light = sysLightDefinitions[lightSample.index];

    unitSquareToSphere(sample.x, sample.y, lightSample.direction, lightSample.pdf);

    // Environment lights do not set the light sample position!
    lightSample.distance = RT_DEFAULT_MAX; // Environment light.

    const float3 &R = lightSample.direction;
    // The seam u == 0.0 == 1.0 is in positive z-axis direction.
    // Compensate for the environment rotation done inside the direct lighting.
    const float u     = (atan2f(R.x, -R.z) + M_PIf) * 0.5f * M_1_PIf + sysEnvironmentRotation; // DAR FIXME Use a light.matrix to rotate the environment.
    const float theta = acosf(-R.y);     // theta == 0.0f is south pole, theta == M_PIf is north pole.
    const float v     = theta * M_1_PIf; // Texture is with origin at lower left, v == 0.0f is south pole.

    float3 texColor = make_float3(1.0f);
    if (light.environmentTextureID != RT_TEXTURE_ID_NULL)
        texColor = make_float3(optix::rtTex2D<float4>(light.environmentTextureID, u, v));
    lightSample.emission = light.emission * texColor;
}

RT_CALLABLE_PROGRAM void sample_directional_light(float3 const& point, const float2 sample, LightSample& lightSample)
{
    LightDefinition light = sysLightDefinitions[lightSample.index];

    lightSample.distance = 1e9;

    lightSample.position = point + lightSample.distance * light.direction;
    lightSample.direction = light.direction;
    lightSample.pdf = 1;

    lightSample.emission = light.emission;
}