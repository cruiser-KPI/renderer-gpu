#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../core/perraydata.h"
#include "../core/lightdata.h"
#include "../math/basic.h"

rtBuffer<LightDefinition> sysLightDefinitions;

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );
rtDeclareVariable(PerRayData, thePrd, rtPayload, );

RT_PROGRAM void miss_gradient()
{
    LightDefinition light = sysLightDefinitions[0];

    // If the last surface intersection was a diffuse which was directly lit with multiple importance sampling,
    // then calculate light emission with multiple importance sampling as well.

    const float envRotation = light.direction.z * 0.5f;

    const float3 R = theRay.direction;
    // The seam u == 0.0 == 1.0 is in positive z-axis direction.
    // Compensate for the environment rotation done inside the direct lighting.
    const float u     = (atan2f(R.x, -R.z) + M_PIf) * 0.5f * M_1_PIf + envRotation; // DAR FIXME Use a light.matrix to rotate the environment.
    const float theta = acosf(-R.y);     // theta == 0.0f is south pole, theta == M_PIf is north pole.
    const float v     = 1 - theta * M_1_PIf; // Texture is with origin at lower left, v == 0.0f is south pole.

    float3 texColor = make_float3(1.0f);
    if (light.environmentTextureID != RT_TEXTURE_ID_NULL)
        texColor = make_float3(optix::rtTex2D<float4>(light.environmentTextureID, light.textureScale * u, light.textureScale * v));

    const float weightMIS = (thePrd.flags & FLAG_DIFFUSE) ? powerHeuristic(thePrd.pdf, 0.25f * M_1_PIf) : 1.0f;
    thePrd.radiance = make_float3(weightMIS) * light.emission * texColor;

    //TODO proper importance sampling of environment light source

    thePrd.flags |= FLAG_TERMINATE;
}