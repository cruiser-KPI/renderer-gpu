#include "hip/hip_runtime.h"


#include <optix.h>
#include <optixu/optixu_aabb_namespace.h>
#include <optixu/optixu_math_namespace.h>

#include "../core/vertexattributes.h"

rtBuffer<VertexAttributes> attributesBuffer;

RT_PROGRAM void triangle_bbox(int primitiveIndex, float result[6])
{
    const float3 v0 = attributesBuffer[3*primitiveIndex  ].vertex;
    const float3 v1 = attributesBuffer[3*primitiveIndex+1].vertex;
    const float3 v2 = attributesBuffer[3*primitiveIndex+2].vertex;

    const float area = optix::length(optix::cross(v1 - v0, v2 - v0));

    optix::Aabb *aabb = (optix::Aabb *) result;

    if (0.0f < area && !isinf(area))
    {
        aabb->m_min = fminf(fminf(v0, v1), v2);
        aabb->m_max = fmaxf(fmaxf(v0, v1), v2);
    }
    else
    {
        aabb->invalidate();
    }
}