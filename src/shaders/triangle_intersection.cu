#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../core/vertexattributes.h"
#include "../math/basic.h"

rtBuffer<VertexAttributes> attributesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

RT_PROGRAM void triangle_intersection(int primitiveIndex)
{
    VertexAttributes const& a0 = attributesBuffer[3*primitiveIndex  ];
    VertexAttributes const& a1 = attributesBuffer[3*primitiveIndex+1];
    VertexAttributes const& a2 = attributesBuffer[3*primitiveIndex+2];

    float3 n;
    float  t;
    float  beta;
    float  gamma;

    if (intersect_triangle(theRay, a0.vertex, a1.vertex, a2.vertex, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            // Barycentric interpolation:
            const float alpha = 1.0f - beta - gamma;

            // Note: No normalization on the TBN attributes here for performance reasons.
            //       It's done after the transformation into world space anyway.
            varGeoNormal = n;

            if (isNull(a0.tangent)){
                float x1 = a1.vertex.x - a0.vertex.x;
                float x2 = a2.vertex.x - a0.vertex.x;
                float y1 = a1.vertex.y - a0.vertex.y;
                float y2 = a2.vertex.y - a0.vertex.y;
                float z1 = a1.vertex.z - a0.vertex.z;
                float z2 = a2.vertex.z - a0.vertex.z;

                float s1 = a1.texcoord.x - a0.texcoord.x;
                float s2 = a2.texcoord.x - a0.texcoord.x;
                float t1 = a1.texcoord.y - a0.texcoord.y;
                float t2 = a2.texcoord.y - a0.texcoord.y;

                float r = 1.f / (s1 * t2 - s2 * t1);
                varTangent = make_float3((t2 * x1 - t1 * x2) * r, (t2 * y1 - t1 * y2) * r,
                              (t2 * z1 - t1 * z2) * r);
//                float3 tdir((s1 * x2 - s2 * x1) * r, (s1 * y2 - s2 * y1) * r,
//                              (s1 * z2 - s2 * z1) * r);
            }
            else
                varTangent = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;

            if (isNull(a0.normal))
                varNormal = varGeoNormal;
            else
                varNormal = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;

            varTexCoord = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;

            rtReportIntersection(0);
        }
    }
}