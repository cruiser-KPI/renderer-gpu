#include "hip/hip_runtime.h"

#include <optix.h>
#include <optixu/optixu_math_namespace.h>

#include "../core/vertexattributes.h"

rtBuffer<VertexAttributes> attributesBuffer;

// Attributes.
rtDeclareVariable(optix::float3, varGeoNormal, attribute GEO_NORMAL, );
rtDeclareVariable(optix::float3, varTangent,   attribute TANGENT, );
rtDeclareVariable(optix::float3, varNormal,    attribute NORMAL, );
rtDeclareVariable(optix::float3, varTexCoord,  attribute TEXCOORD, );

rtDeclareVariable(optix::Ray, theRay, rtCurrentRay, );

// Intersection routine for indexed interleaved triangle data.
RT_PROGRAM void triangle_intersection(int primitiveIndex)
{
    VertexAttributes const& a0 = attributesBuffer[3*primitiveIndex  ];
    VertexAttributes const& a1 = attributesBuffer[3*primitiveIndex+1];
    VertexAttributes const& a2 = attributesBuffer[3*primitiveIndex+2];

    const float3 v0 = a0.vertex;
    const float3 v1 = a1.vertex;
    const float3 v2 = a2.vertex;

    float3 n;
    float  t;
    float  beta;
    float  gamma;

    if (intersect_triangle(theRay, v0, v1, v2, n, t, beta, gamma))
    {
        if (rtPotentialIntersection(t))
        {
            // Barycentric interpolation:
            const float alpha = 1.0f - beta - gamma;

            // Note: No normalization on the TBN attributes here for performance reasons.
            //       It's done after the transformation into world space anyway.
            varGeoNormal      = n;
            varTangent        = a0.tangent  * alpha + a1.tangent  * beta + a2.tangent  * gamma;
            varNormal         = a0.normal   * alpha + a1.normal   * beta + a2.normal   * gamma;
            varTexCoord       = a0.texcoord * alpha + a1.texcoord * beta + a2.texcoord * gamma;

            rtReportIntersection(0);
        }
    }
}